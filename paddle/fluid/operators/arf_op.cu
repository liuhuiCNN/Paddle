#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include <vector>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/operators/prelu_op.h"
#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define CUDA_NUM_THREADS 1024

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename DeviceContext, typename T>
class CUDAARFKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<Tensor>("Input");
    auto* dinput = context.Output<Tensor>(framework::GradVarName("Input"));
    auto* dout = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<Tensor>("Indices");


    const T* input_ptr = input->data<T>();
    const T* dout_ptr = dout->data<T>();


    // input size
    auto input_dims = input->dims();;
    int nOutputPlane = input_dims[0];
    int nInputPlane = input_dims[1];
    int nOrientation = input_dims[2];
    int kH = input_dims[4];
    int kW = input_dims[5];
    nOutputPlane = nOutputPlane;
    nInputPlane = nInputPlane;
    nOrientation = nOrientation;
    kH = kH;
    kW = kW;

    VLOG(4) << "nOutputPlane:" << nOutputPlane << ", nInputPlane:" << nInputPlane
            << "nOrientation:" << nOrientation << "kH" << kH << "kW" << kW;

  }
};


template <typename T>
__global__ void ARFOpGradKernel(const T* x_ptr, const T* alpha_ptr,
                                  const T* dy_ptr, T* dx_ptr, T* dalpha_ptr,
                                  size_t channel_num, size_t plane_size,
                                  size_t spatial_size, size_t numel,
                                  PRELU_MODE mode) {
  CUDA_KERNEL_LOOP(index, numel) {
    T scale;

    T x = x_ptr[index];
    T dy = dy_ptr[index];
    if (dx_ptr != nullptr) dx_ptr[index] = (x > 0) ? dy : scale * dy;
    if (dalpha_ptr != nullptr) dalpha_ptr[index] = (x > 0) ? 0 : x * dy;
  }
}

template <typename T>
class ARFOpGradFunctor {
 public:
  void operator()(hipStream_t stream, const T* x, const T* alpha, const T* dy,
                  T* dx, T* dalpha, const framework::DDim& input_dims,
                  PRELU_MODE mode) {
    size_t numel = 1;
    for (size_t i = 0; i < input_dims.size(); ++i) {
      numel *= input_dims[i];
    }
    size_t plane_size = numel / input_dims[0] / input_dims[1];
    size_t spatial_size = numel / input_dims[0];

    PReluOpGradKernel<
        T><<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
        x, alpha, dy, dx, dalpha, input_dims[1], plane_size, spatial_size,
        numel, mode);
  }
};

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename DeviceContext, typename T>
class CUDAARFGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* dalpha = context.Output<Tensor>(framework::GradVarName("Alpha"));

    const T* x_ptr = x->data<T>();
    const T* alpha_ptr = alpha->data<T>();
    const T* dy_ptr = dy->data<T>();
    T* dx_ptr = dx ? dx->mutable_data<T>(context.GetPlace()) : nullptr;
    T* dalpha_ptr =
        dalpha ? dalpha->mutable_data<T>(context.GetPlace()) : nullptr;

    if (!dx && !dalpha) return;

    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);
    auto stream = context.cuda_device_context().stream();

    T* dalpha_tmp_ptr;
    Tensor dalpha_tmp;
    if (dalpha_ptr == nullptr) {
      dalpha_tmp_ptr = dalpha_ptr;
    } else {
      auto& dev_ctx = context.template device_context<DeviceContext>();
      dalpha_tmp = context.AllocateTmpTensor<T, DeviceContext>(dim, dev_ctx);
      dalpha_tmp_ptr = dalpha_tmp.mutable_data<T>(context.GetPlace());
    }

    PRELU_MODE m;
    if (mode == "element") {
      m = Element;
    } else if (mode == "channel") {
      m = Channel;
    } else {
      m = Scalar;
    }
    PreluOpGradFunctor<T> prelu_grad;
    prelu_grad(stream, x_ptr, alpha_ptr, dy_ptr, dx_ptr, dalpha_tmp_ptr, dim,
               m);

    if (dalpha_tmp_ptr == nullptr) return;

    std::vector<int> reduce_dims;
    for (size_t i = 0; i < dim.size(); i++) {
      if (mode == "channel" && i == 1) continue;
      if (mode == "element" && i != 0) continue;
      reduce_dims.push_back(i);
    }

    TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
        dalpha_tmp, dalpha, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        IdentityFunctor<T>(), stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    arf, ops::CUDAARFKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAARFKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    arf_grad,
    ops::CUDAARFGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAARFGradKernel<paddle::platform::CUDADeviceContext, double>);
